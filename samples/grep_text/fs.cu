/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <unistd.h>

#include "fs_constants.h"
#include "fs_debug.cu.h"
#include "util.cu.h"
#include "cpu_ipc.cu.h"
#include "mallocfree.cu.h"
#include "fs_structures.cu.h"
#include "timer.h"
#include "hash_table.cu.h"
#include "swapper.cu.h"
#include "fs_calls.cu.h"
#include "fs_initializer.cu.h"

// INCLUDING CODE INLINE - change later
#include "host_loop.h"
//DEBUG
__device__ int countInited[1024];
//


/************GLOBALS********/
// CPU Write-shared memory //
__device__ volatile CPU_IPC_OPEN_Queue* g_cpu_ipcOpenQueue;
__device__ volatile CPU_IPC_RW_Queue* g_cpu_ipcRWQueue; 
//
// manager for rw RPC queue

__device__ volatile GPU_IPC_RW_Manager* g_ipcRWManager;

// Open/Close table
__device__ volatile OTable* g_otable;
// Memory pool
__device__ volatile PPool* g_ppool;
// File table with block pointers
__device__ volatile FTable* g_ftable;

// Radix tree memory pool for rt_nodes
__device__ volatile rt_mempool g_rtree_mempool;

// Hash table with all the previously opened files indexed by their inodes
__device__ volatile hash_table g_closed_ftable;

// file_id uniq counter
__device__ int g_file_id;

//pre close table
__device__ volatile preclose_table* g_preclose_table;



#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>

#define MAIN_FS_FILE
#include "grep_text_gpu.cu"


char*  update_filename(const char* h_filename){
	int n=strlen(h_filename);
	assert(n>0);
	if (n>FILENAME_SIZE) {
		fprintf(stderr,"Filname %s too long, should be only %d symbols including \\0",h_filename,FILENAME_SIZE);
		exit (-1);
	}
	char* d_filename;
	CUDA_SAFE_CALL(hipMalloc(&d_filename,n+1));
	CUDA_SAFE_CALL(hipMemcpy(d_filename, h_filename, n+1,hipMemcpyHostToDevice));
	return d_filename;
}

#include <assert.h>

// size of the output used for data staging
int output_size=FS_BLOCKSIZE;
#define TRIALS 1.0
int main( int argc, char** argv)
{

	
	if(argc<5) {
		fprintf(stderr,"<kernel_iterations> <blocks> <threads> f1 f2 ... f_#files\n");
		return -1;
	}
	int trials=atoi(argv[1]);
	int nblocks=atoi(argv[2]);
	int nthreads=atoi(argv[3]);

	fprintf(stderr," iterations: %d blocks %d threads %d\n",trials, nblocks, nthreads);	

	int num_files=argc-1-3;
	char** d_filenames=NULL;
	

	double total_time=0;
//	int scratch_size=128*1024*1024*4;

for(int i=1;i<trials+1;i++){


	
	volatile GPUGlobals* gpuGlobals;
	initializer(&gpuGlobals);
	
	init_device_app();
	init_app();


	if (num_files>0){
		d_filenames=(char**)malloc(sizeof(char*)*num_files);
		for(int i=0;i<num_files;i++){
			d_filenames[i]=update_filename(argv[i+4]);
			fprintf(stderr,"file -%s\n",argv[i+4]);
		}
	}
	double time_before=_timestamp();
	if (!i) time_before=0;

        grep_text<<<nblocks,nthreads,0,gpuGlobals->streamMgr->kernelStream>>>(d_filenames[0],d_filenames[1],d_filenames[2]);
	
	
	while(true)
	{
		open_loop(gpuGlobals,0);
		rw_loop(gpuGlobals);
		if ( hipErrorNotReady != hipStreamQuery(gpuGlobals->streamMgr->kernelStream)) {
			fprintf(stderr,"kernel is complete\n");
			fprintf(stderr,"Max pending requests: %d\n",max_req);
			fprintf(stderr,"Transfer time: %.3f\n",transfer_time);
			transfer_time=0;
			break;
		}
	}


    hipError_t error = hipDeviceSynchronize();
	double time_after=_timestamp();
	if(!i) time_after=0;
	total_time+=(time_after-time_before);

    //Check for errors and failed asserts in asynchronous kernel launch.
    if(error != hipSuccess )
    {
        printf("Device failed, CUDA error message is: %s\n\n", hipGetErrorString(error));
    }
	

    //PRINT_DEBUG;

	fprintf(stderr,"\n");
	delete gpuGlobals;

	PRINT_MALLOC;
	PRINT_FREE;
	PRINT_PAGE_ALLOC_RETRIES;
	PRINT_LOCKLESS_SUCCESS;
	PRINT_WRONG_FILE_ID;

	PRINT_RT_MALLOC;
	PRINT_RT_FREE;
	PRINT_HT_MISS;
	PRINT_PRECLOSE_PUSH;
	PRINT_PRECLOSE_FETCH;
	PRINT_HT_HIT;
	PRINT_FLUSHED_READ;
	PRINT_FLUSHED_WRITE;
	PRINT_TRY_LOCK_FAILED;


//	hipFree(d_output);	
	hipDeviceReset();
	if(error) break;

}
	if (d_filenames) free(d_filenames);

	double thpt=post_app(total_time,trials);

	fprintf(stderr,"Performance: %.3f usec FS_BLOCKSIZE %d FS_LOGBLOCKSIZE %d\n",total_time/trials,FS_BLOCKSIZE, FS_LOGBLOCKSIZE );
//((double)output_size*(double)nblocks*(double)read_count)/(total_time/TRIALS)/1e3 );
	return 0;
}



