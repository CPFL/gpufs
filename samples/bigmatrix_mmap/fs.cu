#include "hip/hip_runtime.h"
/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <unistd.h>


#include "fs_constants.h"
#include "fs_debug.cu.h"
#include "util.cu.h"
#include "cpu_ipc.cu.h"
#include "mallocfree.cu.h"
#include "fs_structures.cu.h"
#include "timer.h"
#include "hash_table.cu.h"
#include "swapper.cu.h"
#include "fs_calls.cu.h"

/************GLOBALS********/
// CPU Write-shared memory //
__device__ volatile CPU_IPC_OPEN_Queue* g_cpu_ipcOpenQueue;
__device__ volatile CPU_IPC_RW_Queue* g_cpu_ipcRWQueue; 
//
// manager for rw RPC queue

__device__ volatile GPU_IPC_RW_Manager* g_ipcRWManager;

// Open/Close table
__device__ volatile OTable* g_otable;
// Memory pool
__device__ volatile PPool* g_ppool;
// File table with block pointers
__device__ volatile FTable* g_ftable;

// Radix tree memory pool for rt_nodes
__device__ volatile rt_mempool g_rtree_mempool;

// Hash table with all the previously opened files indexed by their inodes
__device__ volatile hash_table g_closed_ftable;

// file_id uniq counter
__device__ int g_file_id;

//pre close table
__device__ volatile preclose_table* g_preclose_table;



#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <string.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <math.h>


#include "fs_initializer.cu.h"
// INCLUDING CODE INLINE - change later
#include "host_loop.h"

#define MAIN_FS_FILE
#include "bigmatrix_mmap.cu"

void stdavg(double *avg_time, double *avg_thpt, double* std_time, double *std_thpt, const double* times, const double total_data, int arr_len)
{
	*avg_time=*avg_thpt=*std_time=*std_thpt=0;
	int counter=0;

	for( int i=0;i<arr_len;i++){
		if (times[i]<=0) continue;

		*avg_time+=times[i];
		*avg_thpt+=((double)total_data)/times[i];
		counter++;
	}
	if (counter==0) return;
	*avg_time/=(double)counter;
	*avg_thpt/=(double)counter;

	for( int i=0;i<arr_len;i++){
		if (times[i]<=0) continue;
		*std_time=(times[i]-*avg_time)*(times[i]-*avg_time);
	
		double tmp=(((double)total_data)/times[i])-*avg_thpt;
		*std_thpt=tmp*tmp;
	}
	*std_time/=(double)counter;
	*std_thpt/=(double)counter;

	*std_time=sqrt(*std_time);
	*std_thpt=sqrt(*std_thpt);
	
}


char*  update_filename(const char* h_filename){
	int n=strlen(h_filename);
	assert(n>0);
	if (n>FILENAME_SIZE) {
		fprintf(stderr,"Filname %s too long, should be only %d symbols including \\0",h_filename,FILENAME_SIZE);
		exit (-1);
	}
	char* d_filename;
	CUDA_SAFE_CALL(hipMalloc(&d_filename,n+1));
	CUDA_SAFE_CALL(hipMemcpy(d_filename, h_filename, n+1,hipMemcpyHostToDevice));
	return d_filename;
}

#include <assert.h>

// size of the output used for data staging
int output_size=FS_BLOCKSIZE;

#define MAX_TRIALS (10)
double time_res[MAX_TRIALS];

int main( int argc, char** argv)
{

	
	if(argc<5) {
		fprintf(stderr,"<kernel_iterations> <blocks> <threads> f1 f2 ... f_#files\n");
		return -1;
	}
	int trials=atoi(argv[1]);
	assert(trials<=MAX_TRIALS);
	int nblocks=atoi(argv[2]);
	int nthreads=atoi(argv[3]);

	fprintf(stderr," iterations: %d blocks %d threads %d\n",trials, nblocks, nthreads);	

	int num_files=argc-1-3;
	char** d_filenames=NULL;
	

	double total_time=0;
	size_t total_size;
	
	CUDA_SAFE_CALL(hipSetDevice(2));

	memset(time_res,0,MAX_TRIALS*sizeof(double));
for(int i=0;i<trials+1;i++){


	

	volatile GPUGlobals* gpuGlobals;
	initializer(&gpuGlobals);
	
	init_device_app();
	init_app();


	if (num_files>0){
		d_filenames=(char**)malloc(sizeof(char*)*num_files);
		for(int i=0;i<num_files;i++){
			d_filenames[i]=update_filename(argv[i+4]);
			fprintf(stderr,"file -%s\n",argv[i+4]);
		}
	}
	double time_before=_timestamp();
	if (!i) time_before=0;
	// vector, matrix, out
        bigmatrix_mmap<<<nblocks,nthreads,0,gpuGlobals->streamMgr->kernelStream>>>(d_filenames[0],d_filenames[1],d_filenames[2]);
	
	
	
	int device_num=0;
	while(true)
	{
		open_loop(gpuGlobals,2);
		rw_loop(gpuGlobals);
		if ( hipErrorNotReady != hipStreamQuery(gpuGlobals->streamMgr->kernelStream)) {
			fprintf(stderr,"kernel is complete\n");
			fprintf(stderr,"Max pending requests: %d\n",max_req);
			fprintf(stderr,"Transfer time: %.3f\n",transfer_time);
			transfer_time=0;
			break;
		}
	}


    hipError_t error = hipDeviceSynchronize();
	double time_after=_timestamp();
	if(!i) time_after=0;
	total_time+=(time_after-time_before);
	if (i>0) {time_res[i]=time_after-time_before;
		fprintf(stderr," t-%.3f-us\n",time_res[i]);
	}
	

    //Check for errors and failed asserts in asynchronous kernel launch.
    if(error != hipSuccess )
    {
        printf("Device failed, CUDA error message is: %s\n\n", hipGetErrorString(error));
    }
	

    //PRINT_DEBUG;

	fprintf(stderr,"\n");
	delete gpuGlobals;

	PRINT_MALLOC;
	PRINT_FREE;
	PRINT_PAGE_ALLOC_RETRIES;
	PRINT_LOCKLESS_SUCCESS;
	PRINT_WRONG_FILE_ID;

	PRINT_RT_MALLOC;
	PRINT_RT_FREE;
	PRINT_HT_MISS;
	PRINT_PRECLOSE_PUSH;
	PRINT_PRECLOSE_FETCH;
	PRINT_HT_HIT;
	PRINT_FLUSHED_READ;
	PRINT_FLUSHED_WRITE;
	PRINT_TRY_LOCK_FAILED;


//	hipFree(d_output);	
	hipDeviceReset();
	if(error) break;

	

}

	

	if (d_filenames) free(d_filenames);

	double thpt=post_app(total_time,trials);
	struct stat s1,s2,s3;
	if (stat(argv[4],&s1)) perror("stat failed");
	if (stat(argv[5],&s2)) perror("stat failed");
	if (stat(argv[6],&s3)) perror("stat failed");
	total_size=s1.st_size+s2.st_size+s3.st_size;
	double d_size=total_size/1024.0/1024.0/1024.0;

	
	double avg_time,avg_thpt,std_time,std_thpt;
	
	stdavg(&avg_time,&avg_thpt, &std_time, &std_thpt, time_res, d_size, MAX_TRIALS);

	fprintf(stderr,"Performance: %.3f usec +/- %.3f, %.3f GB,  %.3f GB/s +/- %.3f, FS_BLOCKSIZE %d FS_LOGBLOCKSIZE %d\n",avg_time,std_time, d_size,
				avg_thpt*1e6,std_thpt*1e6,FS_BLOCKSIZE, FS_LOGBLOCKSIZE );
//((double)output_size*(double)nblocks*(double)read_count)/(total_time/TRIALS)/1e3 );
	return 0;
}



