#include "hip/hip_runtime.h"
/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/

/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/


#ifndef MALLOCFREE_CU
#define MALLOCFREE_CU

#include "fs_constants.h"
#include "fs_debug.cu.h"
#include "fs_globals.cu.h"
#include "util.cu.h"
#include "mallocfree.cu.h"
#include "hashMap.cu.h"
#include <assert.h>


// MUST be called from a single thread
DEBUG_NOINLINE __device__  void PPool::init_thread(volatile Page* _storage) volatile
{
	rawStorage=_storage;
	head=0;
	tail=0;
	swapLock=0;
	size=PPOOL_FRAMES;

	for(int i=0;i<PPOOL_FRAMES;i++)
	{
		frames[i].init_thread(&rawStorage[i],i);
		freeList[i]=i;
	}
}
	
DEBUG_NOINLINE __device__ volatile PFrame* PPool::allocPage() volatile
{
	PAGE_ALLOC_START_WARP

	int oldSize = atomicSub( (int*) &size, 1 );

	if( 0 < oldSize )
	{
		uint freeLoc = atomicInc( (uint*) &head, PPOOL_FRAMES - 1 );
		volatile PFrame* pFrame = &( frames[freeList[freeLoc]] );

		GPU_ASSERT( freeList[freeLoc] == pFrame->rs_offset );

		PAGE_ALLOC_STOP_WARP

		return pFrame;
	}

	// else, we are out of memory
	if( MUTEX_TRY_LOCK(swapLock) )
	{
		// swap
		uint numSwapped = 0;
		int numRetries = 0;

		while( NUM_PAGES_SWAPOUT > numSwapped )
		{
			volatile PFrame* cand = &( frames[freeList[tail]] );

			// Try to remove from the hash
			bool removed = false;

			if( cand->dirty == 0 && cand->dirtyCounter == 0 )
			{
				removed = g_hashMap->removePFrame( cand );
			}

			if( removed )
			{
				freePage( cand );
				numSwapped++;
				continue;
			}

			// else
			// Search for another one
			// In this case we will need to swap the element in tail to prevent loosing it later
			uint candLoc = ( tail + 1 ) % PPOOL_FRAMES;

			while( (NUM_SWAP_RETRIES > numRetries) || (0 == numSwapped) )
			{
				cand = &( frames[freeList[candLoc]] );

				bool removed = false;

				if( cand->dirty == 0 && cand->dirtyCounter == 0 )
				{
					removed = g_hashMap->removePFrame( cand );
				}

				if( removed )
				{
					// swap tail and current location
					uint t = freeList[tail];
					freeList[tail] = freeList[candLoc];
					freeList[candLoc] = t;

					threadfence();

					freePage( cand );
					numSwapped++;
					break;
				}

				candLoc = ( candLoc + 1 ) % PPOOL_FRAMES;
				numRetries++;

				GPU_ASSERT(numRetries < (PPOOL_FRAMES / 2))
			}

			if( NUM_SWAP_RETRIES <= numRetries  )
			{
				break;
			}
		}

		GPU_ASSERT( numSwapped > 0 );

		uint freeLoc = atomicInc( (uint*) &head, PPOOL_FRAMES - 1 );
		volatile PFrame* pFrame = &( frames[freeList[freeLoc]] );

		GPU_ASSERT( freeList[freeLoc] == pFrame->rs_offset );

		PAGE_ALLOC_STOP_WARP

		atomicAdd( (int*) &size, numSwapped );

		MUTEX_UNLOCK( swapLock );

		return pFrame;
	}
	else
	{
		// Not enough memory, and someone is already swapping
		// Abort
		atomicAdd( (int*) &size, 1 );
		return NULL;
	}
}

DEBUG_NOINLINE __device__ void PPool::freePage(volatile PFrame* frame) volatile
{
	GPU_ASSERT( freeList[tail] == frame->rs_offset );

	frame->clean();
	freeList[tail] = frame->rs_offset;
	tail = ( tail + 1 ) % PPOOL_FRAMES;
	threadfence();
}

DEBUG_NOINLINE __device__ bool PPool::tryLockSwapper() volatile
{
	return MUTEX_TRY_LOCK(swapLock);
}

DEBUG_NOINLINE __device__ void PPool::lockSwapper() volatile
{
	MUTEX_LOCK(swapLock);
}

DEBUG_NOINLINE __device__ void PPool::unlockSwapper() volatile
{
	MUTEX_UNLOCK(swapLock);
}

#endif
